#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define RANGE 17.78
// Constants for blocks and threads per block
#define BLOCKS 4
#define THREADS_PER_BLOCK 500

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void vecGPU(float *ad, float *bd, float *cd, int n);
/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; //temp is just another copy of C
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	
	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device  
	*/
	int size = n * sizeof(float);

	//Allocate the arrays in the device

	if (hipMalloc((void **)&ad, size) != hipSuccess)
	{
		printf("Error allocating array ad of %d elements on device\n", n);
		exit(1);
	}

	if (hipMalloc((void **)&bd, size) != hipSuccess)
	{
		printf("Error allocating array bd of %d elements on device\n", n);
		exit(1);
	}

	if (hipMalloc((void **)&cd, size) != hipSuccess)
	{
		printf("Error allocating array cd of %d elements on device\n", n);
		exit(1);
	}

	//Copy a, b, and c to the device

	if (hipMemcpy(ad, a, size, hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying array a from host to device\n");
        exit(1);
    }

	if (hipMemcpy(bd, b, size, hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying array b from host to device\n");
        exit(1);
    }

	if (hipMemcpy(cd, c, size, hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying array c from host to device\n");
        exit(1);
    }

	start = clock();

	/* TODO: 	
		3. write the kernel, call it: vecGPU
		4. call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to decide about the number of threads, blocks, etc and their geometry.
	*/

	int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid = BLOCKS;

	//Launch the kernel
	vecGPU<<<blocksPerGrid, threadsPerBlock>>>(ad, bd, cd, n);

	//Block host till device is done
	hipDeviceSynchronize();

	end = clock();

	/* TODO: 
		5. bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/
	
	//Copy result back to host
	if (hipMemcpy(c, cd, size, hipMemcpyDeviceToHost) != hipSuccess) {
        printf("Error copying array c from device to host\n");
        exit(1);
    }

	//Free the arrays in the device
    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
	
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++)
	  if( fabs(temp[i] - c[i]) >= 0.009) //compare up to the second degit in floating point
		printf("Element %d in the result array does not match the sequential version\n", i);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/

__global__ void vecGPU(float *ad, float *bd, float *cd, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        cd[idx] += ad[idx] * bd[idx];
    }
}